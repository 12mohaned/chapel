#ifdef __cplusplus
extern "C" {
#include <stdio.h>
#include "chplgpu.h"
	void gpuAllocate(void **data, size_t size) {
		hipMalloc( data, size);
		printf("gpuAllocate: Allocated %u bytes at %p\n", (unsigned int)size, data, *data);
	}

	void gpuFree(void **data) {
		printf("Freed data : gpuFree at %p\n", data);
		hipFree( *data);
	}

	void copyGPUtoHost(void **dst, void **src, size_t size) {
		printf("** Copying gpu to host : DST = %p SRC = %p SIZE = %u\n", *dst, *src, (unsigned int)size);
		hipMemcpy( *dst, *src, size, hipMemcpyDeviceToHost);
	}
	
	void copyHostToGPU(void **dst, void **src, size_t size) {
		printf("** Copying host to gpu : DST = %p SRC = %p SIZE = %u\n", *dst, *src, (unsigned int)size);
		hipMemcpy( *dst, *src, size, hipMemcpyHostToDevice);
		
	}
}
#endif
